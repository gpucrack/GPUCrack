#include "hip/hip_runtime.h"
#include "chains.cuh"

__device__ static const unsigned char charset[CHARSET_LENGTH] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 'A',
                                                                 'B', 'C',
                                                                 'D', 'E',
                                                                 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P',
                                                                 'Q', 'R',
                                                                 'S', 'T',
                                                                 'U', 'V', 'W', 'X', 'Y', 'Z', 'a', 'b', 'c',
                                                                 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n',
                                                                 'o', 'p',
                                                                 'q', 'r',
                                                                 's', 't',
                                                                 'u', 'v', 'w', 'x', 'y', 'z'};

__host__ void
generateChains(Password *h_passwords, Digest *h_results, int passwordNumber, int numberOfPass, int numberOfColumn,
               bool save, int theadsPerBlock, bool debug, bool debugKernel) {

    float milliseconds = 0;

    int batchSize = computeBatchSize(numberOfPass, passwordNumber);

    // We send numberOfColumn/2 since one loop of kernel is hashing/reducing at the same time so we need 2x
    // less operations
    chainKernel(passwordNumber, numberOfPass, batchSize, &milliseconds,
                &h_passwords, &h_results, theadsPerBlock,
                numberOfColumn / 2, save, debugKernel);

    if (debug) {
        printf("Total GPU time : %f milliseconds\n", milliseconds);
        printf("Chain rate : %f MC/s\n",
               ((float) (passwordNumber) / (milliseconds / 1000)) / 1000000);
        printf("Column rate : %f MCo/s\n",
               (((float) (passwordNumber) / (milliseconds / 1000)) / 1000000) * (float)(numberOfColumn));
    }
}

__global__ void ntlmChainKernel(Password *passwords, Digest *digests, int chainLength) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i < chainLength; i++) {
        ntlm(&passwords[index], &digests[index]);
        reduceDigest(i, &digests[index], &passwords[index]);
    }
}

__global__ void ntlmChainKernelDebug(Password *passwords, Digest *digests, int chainLength) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i < chainLength; i++) {
        if(index == 0){
            printPassword(&passwords[index]);
            printf(" --> ");
        }
        ntlm(&passwords[index], &digests[index]);
        if (index == 0){
            printDigest(&digests[index]);
            printf(" --> ");
        }
        reduceDigest(i, &digests[index], &passwords[index]);
        if(index == 0){
            printPassword(&passwords[index]);
            printf("\n");
        }
    }
}

__host__ void
chainKernel(int passwordNumber, int numberOfPass, int batchSize, float *milliseconds, Password **h_passwords,
            Digest **h_results, int threadPerBlock, int chainLength, bool save, bool debug) {

    if (save) {
        createFile((char *) "testStart.bin", true);
        writePoint((char *) "testStart.bin", h_passwords, passwordNumber, chainLength, true);
    }

    double program_time_used;
    clock_t program_start, program_end;
    program_start = clock();

    // Device copies for endpoints
    Digest *d_results;
    Password *d_passwords;

    // Measure GPU time
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipStream_t stream1;

    hipStreamCreate(&stream1);

    int chainsRemaining = passwordNumber;
    int currentIndex = 0;

    printf("Generating chains...\n\n");

    // Main loop, we add +1 to be sure to do all the batches in case
    // we have 2.5 for example, it'll be 3 passes
    for (int i = 0; i < numberOfPass; i++) {
        // Temporary variable to measure GPU time inside this loop
        float tempMilli = 0;

        // If the currentIndex to save result is greater than the number of
        // password we must stop
        if (currentIndex >= passwordNumber) break;

        // If we have less than batchSize password to hash, then hash them all
        // but modify the batchSize to avoid index errors
        if (chainsRemaining <= batchSize) batchSize = chainsRemaining;

        // GPU Malloc for the password array, size is batchSize
        hipMalloc(&d_passwords, sizeof(Password) * batchSize);
        hipMalloc(&d_results, sizeof(Digest) * batchSize);

        Password *source = *h_passwords;

        // Device copies
        hipMemcpyAsync(d_passwords, &(source[currentIndex]), sizeof(Password) * batchSize,
                        hipMemcpyHostToDevice, stream1);

        hipEventRecord(start);
        if (debug)
            ntlmChainKernelDebug<<<((batchSize) / threadPerBlock), threadPerBlock, 0, stream1>>>(
                    d_passwords, d_results, chainLength);
        else
            ntlmChainKernel<<<((batchSize) / threadPerBlock), threadPerBlock, 0, stream1>>>(
                    d_passwords, d_results, chainLength);
        hipEventRecord(end);
        hipEventSynchronize(end);

        // Necessary procedure to record time and store the elasped time in
        // tempMilli
        hipEventElapsedTime(&tempMilli, start, end);
        *milliseconds += tempMilli;
        hipEventDestroy(start);
        hipEventDestroy(end);

        // Check for errors during hashKernel execution
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) {
            printf("hashKernel launch failed with error \"%s\".\n",
                   hipGetErrorString(cudaerr));
            exit(1);
        }

        Digest *destination = *h_results;
        // Device to host copy

        hipMemcpyAsync(&(destination[currentIndex]), d_results,
                        sizeof(Digest) * batchSize, hipMemcpyDeviceToHost, stream1);

        Password *destination2 = *h_passwords;
        // Device to host copy

        hipMemcpyAsync(&(destination2[currentIndex]), d_passwords,
                        sizeof(Password) * batchSize, hipMemcpyDeviceToHost, stream1);

        currentIndex += batchSize;
        chainsRemaining -= batchSize;

        // Cleanup before next loop to free memory
        hipFree(d_passwords);
        hipFree(d_results);
    }
    hipStreamDestroy(stream1);

    program_end = clock();
    program_time_used =
            ((double) (program_end - program_start)) / CLOCKS_PER_SEC;
    printf("Total execution time : %f seconds = %f minutes = %f hours\n", program_time_used,
           program_time_used/60, (program_time_used/60)/60);

    if (save) {
        createFile((char *) "testEnd.bin", true);
        writePoint((char *) "testEnd.bin", h_passwords, passwordNumber, chainLength,
                   true);
    }
}