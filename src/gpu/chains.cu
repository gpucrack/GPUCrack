#include "hip/hip_runtime.h"
#include "chains.cuh"

__host__ void
generateChains(Password *h_passwords, unsigned long long passwordNumber, int numberOfPass, int numberOfColumn,
               bool save, int theadsPerBlock, bool debug, bool debugKernel, Digest *h_results, int pwd_length,
               char *start_path, char *end_path, float *totalGPU, int batchNumber) {

    printf("Generating chains...\n");

    float milliseconds = 0;

    unsigned long long batchSize = computeBatchSize(numberOfPass, passwordNumber);

    // We send numberOfColumn/2 since one loop of kernel is hashing/reducing at the same time so we need 2x
    // less operations
    chainKernel(passwordNumber, numberOfPass, batchSize, &milliseconds,
                &h_passwords, theadsPerBlock,
                numberOfColumn, debugKernel, &h_results, pwd_length, start_path, end_path, debugKernel);

    printf("Batch %d done in : %f milliseconds (GPU time).\n", batchNumber, milliseconds);

    *totalGPU = *totalGPU + milliseconds;
}

__global__ void
ntlmChainKernel(Password *passwords, Digest *digests, int chainLength, int pwd_length, unsigned long long domain) {
    unsigned long long index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i < chainLength; i++) {
        ntlm(&passwords[index], &digests[index], pwd_length);
        reduceDigest(i, &digests[index], &passwords[index], pwd_length, domain);
    }
}

__global__ void
ntlmChainKernelDebug(Password *passwords, Digest *digests, int chainLength, int pwd_length, unsigned long long domain) {
    unsigned long long index = blockIdx.x * blockDim.x + threadIdx.x;

    // Trick to for a working print
    Password * password = (Password*) malloc(sizeof(Password));
    Digest * digest = (Digest*) malloc(sizeof(Digest));
    for (int i = 0; i < chainLength; i++) {
        if(index == (1)){
            printf("%d: ", i);
            printPassword(&passwords[index]);
            printf(" --> ");
        }
        ntlm(&passwords[index], &digests[index], pwd_length);
        if (index == (1)){
            printDigest(&digests[index]);
            printf(" --> ");
        }
        reduceDigest(i, &digests[index], &passwords[index], pwd_length, domain);
        if(index == (1)){
            printPassword(&passwords[index]);
            printf("\n");
        }
    }
    free(password);
    free(digest);
}

__host__ void
chainKernel(unsigned long long passwordNumber, int numberOfPass, unsigned long long batchSize, float *milliseconds,
            Password **h_passwords, int threadPerBlock, int chainLength, bool debug, Digest **h_results,
            int pwd_length, char *start_path, char *end_path, bool kernelDebug) {

    unsigned long long domain = (unsigned long long)pow(CHARSET_LENGTH, pwd_length);

    if (debug) printf("Domain : %lld\n", domain);

    // Device copies for endpoints
    Digest *d_results;
    Password *d_passwords;

    // Measure GPU time
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipStream_t stream1;

    hipStreamCreate(&stream1);

    unsigned long long chainsRemaining = passwordNumber;
    unsigned long long currentIndex = 0;

    if (debug) printf("Generating chains...\n\n");

    // Main loop, we add +1 to be sure to do all the batches in case
    // we have 2.5 for example, it'll be 3 passes
    for (int i = 0; i < numberOfPass; i++) {
        // Temporary variable to measure GPU time inside this loop
        float tempMilli = 0;

        // If the currentIndex to save result is greater than the number of
        // password we must stop
        if (currentIndex >= passwordNumber) break;

        // If we have less than batchSize password to hash, then hash them all
        // but modify the batchSize to avoid index errors
        if (chainsRemaining <= batchSize) batchSize = chainsRemaining;

        // GPU Malloc for the password array, size is batchSize
        handleCudaError(hipMalloc(&d_passwords, sizeof(Password) * batchSize));
        handleCudaError(hipMalloc(&d_results, sizeof(Digest) * batchSize));

        Password *source = *h_passwords;

        // Device copies
        handleCudaError(hipMemcpyAsync(d_passwords, &(source[currentIndex]), sizeof(Password) * batchSize,
                        hipMemcpyHostToDevice, stream1));

        hipEventRecord(start);

        if (kernelDebug)
            ntlmChainKernelDebug<<<((unsigned long long)((unsigned long long)(batchSize) / (unsigned long long)threadPerBlock)) + 1, threadPerBlock, 0, stream1>>>(
                    d_passwords, d_results, chainLength, pwd_length, domain);
        else
            ntlmChainKernel<<<((unsigned long long)((unsigned long long)(batchSize) / (unsigned long long)threadPerBlock)) + 1, threadPerBlock, 0, stream1>>>(
                    d_passwords, d_results, chainLength, pwd_length, domain);

        hipEventRecord(end);
        hipEventSynchronize(end);

        // Necessary procedure to record time and store the elasped time in
        // tempMilli
        hipEventElapsedTime(&tempMilli, start, end);
        *milliseconds += tempMilli;
        hipEventDestroy(start);
        hipEventDestroy(end);

        // Check for errors during hashKernel execution
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) {
            printf("hashKernel launch failed with error \"%s\".\n",
                   hipGetErrorString(cudaerr));
            exit(1);
        }

        if (debug){
            Digest *destination = *h_results;
            // Device to host copy

            handleCudaError(hipMemcpyAsync(&(destination[currentIndex]), d_results,
                            sizeof(Digest) * batchSize, hipMemcpyDeviceToHost, stream1));
        }

        Password *destination2 = *h_passwords;
        // Device to host copy
        handleCudaError(hipMemcpyAsync(&(destination2[currentIndex]), d_passwords,
                        sizeof(Password) * batchSize, hipMemcpyDeviceToHost, stream1));

        currentIndex += batchSize;
        chainsRemaining -= batchSize;

        // Cleanup before next loop to free memory
        hipFree(d_passwords);
        hipFree(d_results);
    }
    hipStreamDestroy(stream1);
}