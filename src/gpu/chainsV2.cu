#include "hip/hip_runtime.h"
#include "chainsV2.cuh"

__host__ int createChain() {
    double program_time_used;
    clock_t program_start, program_end;
    program_start = clock();

    int t = 0;

    // Measure time
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);

    ntlm_chain_kernel2<<<DEFAULT_PASSWORD_NUMBER / THREAD_PER_BLOCK, THREAD_PER_BLOCK>>>(
            t);
    hipEventRecord(end);

    // Check for errors during kernel execution
    hipError_t hipError_t = hipDeviceSynchronize();
    if (hipError_t != hipSuccess) {
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(hipError_t));
        return 1;
    }

    printf("KERNEL DONE in %f seconds\n",
           (double) (clock() - program_start) / CLOCKS_PER_SEC);

    hipEventSynchronize(end);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);

    // Copy back the device result array to host result array
    hipMemcpy(&table, d_table, sizeof(RainbowTable), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_table);

    program_end = clock();
    program_time_used =
            ((double) (program_end - program_start)) / CLOCKS_PER_SEC;
    printf("TOTAL EXECUTION TIME : %f seconds\n", program_time_used);
}

__global__ void ntlm_chain_kernel2(int t) {

}

int main() {
    return createChain();
}