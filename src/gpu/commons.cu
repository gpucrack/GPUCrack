#include "hip/hip_runtime.h"
#include "commons.cuh"

__host__ void generatePasswords(Password ** result, int passwordNumber) {

    hipError_t status = hipHostMalloc(result, passwordNumber * sizeof(Password), hipHostMallocDefault);
    if (status != hipSuccess)
        printf("Error allocating pinned host memory\n");

    generateNewPasswords(result, passwordNumber);
}

__host__ void generateNewPasswords(Password ** result, int passwordNumber) {

    char charSet[62] = {'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's',
                        't', 'u', 'v', 'w', 'x',
                        'y', 'z', '1', '2', '3', '4', '5', '6', '7', '8', '9', '0', 'A', 'B', 'C', 'D', 'E', 'F', 'G',
                        'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z'};

    std::random_device rd; // obtain a random number from hardware
    std::mt19937 gen(rd()); // seed the generator
    std::uniform_int_distribution<> distr(0, 61); // define the range

    printf("\n==========GENERATING PASSWORDS==========\n");
    // Generate all passwords
    for (int j = 0; j < passwordNumber; j++) {
        // Generate one password
        for (int i=0; i<PASSWORD_LENGTH; i++) {
            (*result)[j].bytes[i] = charSet[distr(gen)];
        }
    }
    printf("DONE, %d PASSWORDS GENERATED\n", passwordNumber);
}

// Returns the number of batch that we need to do
__host__ int memoryAnalysis(int passwordNumber) {

    printf("\n==========MEMORY ANALYSIS==========\n");

    // Checking if THREAD_PER_BLOCK is a power of 2 because we will have memory problems otherwise
    if ((ceil(log2(THREAD_PER_BLOCK)) != floor(log2(THREAD_PER_BLOCK)))) {
        printf("THREAD PER BLOCK VALUE IS NOT A POWER OF 2 !\n");
        exit(1);
    }

    size_t freeMem;
    size_t totalMem;
    hipError_t mem = hipMemGetInfo(&freeMem, &totalMem);

    // Just to keep a little of memory, just in case
    freeMem -= 500000000;

    // Checking errors on memory detection
    if (mem != hipSuccess) {
        printf("memory check failed with error \"%s\".\n",
               hipGetErrorString(mem));
        exit(1);
    }

    printf("MEMORY AVAILABLE : %ld Megabytes\n", (freeMem / 1000000));

    // Computing memory used by password and result array
    size_t memResult = sizeof(Digest) * passwordNumber;
    size_t memPasswords = sizeof(Password) * passwordNumber;
    size_t memUsed = memPasswords + memResult;

    printf("MEMORY USED BY RESULT ARRAY : %ld Megabytes\n",
           (memResult / 1000000));
    printf("MEMORY USED BY PASSWORD ARRAY : %ld Megabytes\n",
           (memPasswords / 1000000));

    printf("THIS MUCH MEMORY WILL BE USED : %ld Megabytes\n",
           (memUsed / 1000000));

    // We need to determine how many batch we'll do to hash all passwords
    // We need to compute the batch size as well
    auto numberOfPass = (double) ((double) memUsed / (double) freeMem);
    if (numberOfPass < 1) return 1;

    numberOfPass += 0.5;

    int finalNumberOfPass = (int) numberOfPass;
    if ((finalNumberOfPass % 2) != 0) finalNumberOfPass++;

    printf("NUMBER OF PASS : %d\n", finalNumberOfPass);

    return finalNumberOfPass;
}

__host__ int computeBatchSize(int numberOfPass, int passwordNumber) {
    int batchSize;

    if (numberOfPass > 1)
        batchSize = (passwordNumber / numberOfPass);
        // If we have less than 1 round then the batch size is the number of
        // passwords
    else
        batchSize = passwordNumber;

    return batchSize;
}

__host__ void initEmptyArrays(Password ** passwords, Digest ** results, int passwordNumber) {

    hipError_t status = hipHostMalloc(passwords, passwordNumber * sizeof(Password), hipHostMallocDefault);
    if (status != hipSuccess)
        printf("Error allocating pinned host memory\n");

    status = hipHostMalloc(results, passwordNumber * sizeof(Digest), hipHostMallocDefault);
    if (status != hipSuccess)
        printf("Error allocating pinned host memory\n");

}

__host__ void initArrays(Password ** passwords, Digest ** results, int passwordNumber) {

    generatePasswords(passwords, passwordNumber);

    hipError_t status = hipHostMalloc(results, passwordNumber * sizeof(Digest), hipHostMallocDefault);
    if (status != hipSuccess)
        printf("Error allocating pinned host memory\n");

}

__host__ void hashKernel(const int numberOfPass, int batchSize,
                         float *milliseconds, const clock_t *program_start,
                         Digest **h_results, Password **h_passwords, int passwordNumber,
                         int threadPerBlock) {

    // Device copies
    Digest *d_results;
    Password *d_passwords;

    // Measure GPU time
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipStream_t stream1;

    hipStreamCreate(&stream1);

    int passwordRemaining = passwordNumber;
    int currentIndex = 0;

    // Main loop, we add +1 to be sure to do all the batches in case
    // we have 2.5 for example, it'll be 3 passes
    for (int i = 0; i<numberOfPass; i++) {
        // Temporary variable to measure GPU time inside this loop
        float tempMilli = 0;

        // If the currentIndex to save result is greater than the number of
        // password we must stop
        if (currentIndex >= passwordNumber) break;

        // If we have less than batchSize password to hash, then hash them all
        // but modify the batchSize to avoid index errors
        if (passwordRemaining <= batchSize) batchSize = passwordRemaining;

        // GPU Malloc for the password array, size is batchSize
        hipMalloc(&d_passwords, sizeof(Password) * batchSize);
        hipMalloc(&d_results, sizeof(Digest) * batchSize);

        Password *source = *h_passwords;

        // Device copies
        hipMemcpyAsync(d_passwords, &(source[currentIndex]), sizeof(Password) * batchSize,
                   hipMemcpyHostToDevice, stream1);

        hipEventRecord(start);
        ntlm_kernel<<<((batchSize) / threadPerBlock), threadPerBlock, 0, stream1>>>(
                d_passwords, d_results);
        hipEventRecord(end);
        hipEventSynchronize(end);

        // Necessary procedure to record time and store the elasped time in
        // tempMilli
        hipEventElapsedTime(&tempMilli, start, end);
        *milliseconds += tempMilli;
        hipEventDestroy(start);
        hipEventDestroy(end);

        // Check for errors during hashKernel execution
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) {
            printf("hashKernel launch failed with error \"%s\".\n",
                   hipGetErrorString(cudaerr));
            exit(1);
        }

        Digest *destination = *h_results;
        // Device to host copy

        hipMemcpyAsync(&(destination[currentIndex]), d_results,
                   sizeof(Digest) * batchSize, hipMemcpyDeviceToHost, stream1);

        currentIndex += batchSize;
        passwordRemaining -= batchSize;

        // Cleanup before next loop to free memory
        hipFree(d_passwords);
        hipFree(d_results);
    }
}

__host__ void chainKernel(int passwordNumber, int numberOfPass, int batchSize, float *milliseconds,
                          Password ** h_passwords, Digest ** h_results, int threadPerBlock,
                          int chainLength) {

    // Device copies for endpoints
    Digest *d_results;
    Password *d_passwords;

    // Measure GPU time
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipStream_t stream1;

    hipStreamCreate(&stream1);

    int chainsRemaining = passwordNumber;
    int currentIndex = 0;

    // Main loop, we add +1 to be sure to do all the batches in case
    // we have 2.5 for example, it'll be 3 passes
    for (int i = 0; i<numberOfPass; i++) {
        // Temporary variable to measure GPU time inside this loop
        float tempMilli = 0;

        // If the currentIndex to save result is greater than the number of
        // password we must stop
        if (currentIndex >= passwordNumber) break;

        // If we have less than batchSize password to hash, then hash them all
        // but modify the batchSize to avoid index errors
        if (chainsRemaining <= batchSize) batchSize = chainsRemaining;

        // GPU Malloc for the password array, size is batchSize
        hipMalloc(&d_passwords, sizeof(Password) * batchSize);
        hipMalloc(&d_results, sizeof(Digest) * batchSize);

        Password *source = *h_passwords;

        // Device copies
        hipMemcpyAsync(d_passwords, &(source[currentIndex]), sizeof(Password) * batchSize,
                        hipMemcpyHostToDevice, stream1);

        hipEventRecord(start);
        ntlm_chain_kernel2<<<((batchSize) / threadPerBlock), threadPerBlock, 0, stream1>>>(
                d_passwords, d_results, chainLength);
        hipEventRecord(end);
        hipEventSynchronize(end);

        // Necessary procedure to record time and store the elasped time in
        // tempMilli
        hipEventElapsedTime(&tempMilli, start, end);
        *milliseconds += tempMilli;
        hipEventDestroy(start);
        hipEventDestroy(end);

        // Check for errors during hashKernel execution
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) {
            printf("hashKernel launch failed with error \"%s\".\n",
                   hipGetErrorString(cudaerr));
            exit(1);
        }

        Digest *destination = *h_results;
        // Device to host copy

        hipMemcpyAsync(&(destination[currentIndex]), d_results,
                        sizeof(Digest) * batchSize, hipMemcpyDeviceToHost, stream1);

        currentIndex += batchSize;
        chainsRemaining -= batchSize;

        // Cleanup before next loop to free memory
        hipFree(d_passwords);
        hipFree(d_results);
    }
}

__host__ void printDigest(Digest * dig) {

    for(unsigned char byte : dig->bytes){
        printf("%X02", byte);
    }

    printf("\n");
}

__host__ void printPassword(Password * pwd) {
    for(unsigned char byte : pwd->bytes){
        printf("%c", byte);
    }
    printf("\n");
}