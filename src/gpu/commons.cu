#include "hip/hip_runtime.h"
#include "commons.cuh"

__host__ void generatePasswords(Password ** result, int passwordNumber) {

    hipError_t status = hipHostMalloc(result, passwordNumber * sizeof(Password), hipHostMallocDefault);
    if (status != hipSuccess)
        printf("Error allocating pinned host memory\n");

    generateNewPasswords(result, passwordNumber);
}

__host__ void generateNewPasswords(Password ** result, int passwordNumber) {

    char charSet[62] = {'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's',
                        't', 'u', 'v', 'w', 'x',
                        'y', 'z', '1', '2', '3', '4', '5', '6', '7', '8', '9', '0', 'A', 'B', 'C', 'D', 'E', 'F', 'G',
                        'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z'};

    std::random_device rd; // obtain a random number from hardware
    std::mt19937 gen(rd()); // seed the generator
    std::uniform_int_distribution<> distr(0, 61); // define the range

    printf("\n==========GENERATING PASSWORDS==========\n");
    // Generate all passwords
    for (int j = 0; j < passwordNumber; j++) {
        // Generate one password
        for (int i=0; i<PASSWORD_LENGTH; i++) {
            (*result)[j].bytes[i] = charSet[distr(gen)];
        }
    }
    printf("DONE, %d PASSWORDS GENERATED\n", passwordNumber);
}

// Returns the number of batch that we need to do
__host__ int memoryAnalysis(int passwordNumber) {

    printf("\n==========MEMORY ANALYSIS==========\n");

    // Checking if THREAD_PER_BLOCK is a power of 2 because we will have memory problems otherwise
    if ((ceil(log2(THREAD_PER_BLOCK)) != floor(log2(THREAD_PER_BLOCK)))) {
        printf("THREAD PER BLOCK VALUE IS NOT A POWER OF 2 !\n");
        exit(1);
    }

    size_t freeMem;
    size_t totalMem;
    hipError_t mem = hipMemGetInfo(&freeMem, &totalMem);

    // Just to keep a little of memory, just in case
    freeMem -= 500000000;

    // Checking errors on memory detection
    if (mem != hipSuccess) {
        printf("memory check failed with error \"%s\".\n",
               hipGetErrorString(mem));
        exit(1);
    }

    printf("MEMORY AVAILABLE : %ld Megabytes\n", (freeMem / 1000000));

    // Computing memory used by password and result array
    size_t memResult = sizeof(Digest) * passwordNumber;
    size_t memPasswords = sizeof(Password) * passwordNumber;
    size_t memUsed = memPasswords + memResult;

    printf("MEMORY USED BY RESULT ARRAY : %ld Megabytes\n",
           (memResult / 1000000));
    printf("MEMORY USED BY PASSWORD ARRAY : %ld Megabytes\n",
           (memPasswords / 1000000));

    printf("THIS MUCH MEMORY WILL BE USED : %ld Megabytes\n",
           (memUsed / 1000000));

    // We need to determine how many batch we'll do to hash all passwords
    // We need to compute the batch size as well
    auto numberOfPass = (double) ((double) memUsed / (double) freeMem);
    if (numberOfPass < 1) return 1;

    numberOfPass += 0.5;

    int finalNumberOfPass = (int) numberOfPass;
    if ((finalNumberOfPass % 2) != 0) finalNumberOfPass++;

    printf("NUMBER OF PASS : %d\n", finalNumberOfPass);

    return finalNumberOfPass;
}

__host__ int computeBatchSize(int numberOfPass, int passwordNumber) {
    int batchSize;

    if (numberOfPass > 1)
        batchSize = (passwordNumber / numberOfPass);
        // If we have less than 1 round then the batch size is the number of
        // passwords
    else
        batchSize = passwordNumber;

    return batchSize;
}

__host__ void initEmptyArrays(Password ** passwords, Digest ** results, int passwordNumber) {

    hipError_t status = hipHostMalloc(passwords, passwordNumber * sizeof(Password), hipHostMallocDefault);
    if (status != hipSuccess)
        printf("Error allocating pinned host memory\n");

    status = hipHostMalloc(results, passwordNumber * sizeof(Digest), hipHostMallocDefault);
    if (status != hipSuccess)
        printf("Error allocating pinned host memory\n");

}

__host__ void initArrays(Password ** passwords, Digest ** results, int passwordNumber) {

    generatePasswords(passwords, passwordNumber);

    hipError_t status = hipHostMalloc(results, passwordNumber * sizeof(Digest), hipHostMallocDefault);
    if (status != hipSuccess)
        printf("Error allocating pinned host memory\n");

}

__device__ __host__ void printDigest(Digest * dig) {

    for(unsigned char byte : dig->bytes){
        printf("%x", byte);
    }

    printf("\n");
}

__device__ __host__ void printPassword(Password * pwd) {
    for(unsigned char byte : pwd->bytes){
        printf("%c", byte);
    }
    printf("\n");
}