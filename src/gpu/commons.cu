#include "hip/hip_runtime.h"
#include "commons.cuh"

__host__ void printSignature() {
    printf("GPUCrack v0.1.1\n"
           "<https://github.com/gpucrack/GPUCrack/>\n\n");
}

__host__ void handleCudaError(hipError_t status) {
    if (status != hipSuccess) {
        const char *errorMessage = hipGetErrorString(status);
        printf("CUDA error: %s.\n", errorMessage);
        exit(1);
    }
}

__host__ void generatePasswords(Password **result, int passwordNumber) {
    handleCudaError(hipHostMalloc(result, passwordNumber * sizeof(Password), hipHostMallocDefault));
    generateNewPasswords2(result, passwordNumber);
}

__host__ void generateNewPasswords2(Password **result, int passwordNumber) {
    char charset[62] = {'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's',
                        't', 'u', 'v', 'w', 'x',
                        'y', 'z', '1', '2', '3', '4', '5', '6', '7', '8', '9', '0', 'A', 'B', 'C', 'D', 'E', 'F', 'G',
                        'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z'};
    char charsetLength = 61;

    for (int j = 0; j < passwordNumber; j++) {
        // Generate one password
        int counter = j;
        for (unsigned char & byte : (*result)[j].bytes) {
            byte = charset[ counter % charsetLength];
            counter /= charsetLength;
        }
    }
}

__host__ void generateNewPasswords(Password **result, int passwordNumber) {

    char charSet[62] = {'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's',
                        't', 'u', 'v', 'w', 'x',
                        'y', 'z', '1', '2', '3', '4', '5', '6', '7', '8', '9', '0', 'A', 'B', 'C', 'D', 'E', 'F', 'G',
                        'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z'};

    std::random_device rd; // obtain a random number from hardware
    std::mt19937 gen(rd()); // seed the generator
    std::uniform_int_distribution<> distr(0, 61); // define the range

    printf("\nGenerating passwords...\n");
    // Generate all passwords
    for (int j = 0; j < passwordNumber; j++) {
        // Generate one password
        for (unsigned char & byte : (*result)[j].bytes) {
            byte = charSet[distr(gen)];
        }
    }
    printf("Done, %d passwords generated\n", passwordNumber);
}

// Returns the number of batch that we need to do
__host__ int memoryAnalysis(int passwordNumber) {

    printf("\n==========GPU MEMORY ANALYSIS==========\n");

    int nDevices;
    hipGetDeviceCount(&nDevices);

    printf("Number of devices: %d\n", nDevices);

    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (MHz): %d\n",
               prop.memoryClockRate/1024);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Total global memory (Gbytes) %.1f\n",(float)(prop.totalGlobalMem)/1024.0/1024.0/1024.0);
        printf("  Shared memory per block (Kbytes) %.1f\n",(float)(prop.sharedMemPerBlock)/1024.0);
        printf("  minor-major: %d-%d\n", prop.minor, prop.major);
        printf("  Warp-size: %d\n", prop.warpSize);
        printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
        printf("  Concurrent computation/communication: %s\n\n",prop.deviceOverlap ? "yes" : "no");
    }

    // Checking if THREAD_PER_BLOCK is a power of 2 because we will have memory problems otherwise
    if ((ceil(log2(THREAD_PER_BLOCK)) != floor(log2(THREAD_PER_BLOCK)))) {
        printf("Thread per block value is not a power of 2 !\n");
        exit(1);
    }


    // Detect available memory
    size_t freeMem;
    size_t totalMem;
    handleCudaError(hipMemGetInfo(&freeMem, &totalMem));

    // Just to keep a little of memory, just in case
    freeMem -= 500000000;

    printf("GPU memory available: %ld Megabytes\n", (freeMem / 1000000));

    // Computing memory used by password and result array
    size_t memResult = sizeof(Digest) * passwordNumber;
    size_t memPasswords = sizeof(Password) * passwordNumber;
    size_t memUsed = memPasswords + memResult;

    printf("Memory used by digest array : %ld Megabytes\n",
           (memResult / 1000000));
    printf("Memory used by password array : %ld Megabytes\n",
           (memPasswords / 1000000));

    printf("This much memory will be used : %ld Megabytes\n\n",
           (memUsed / 1000000));

    if((memUsed / 1000000000) >= getTotalSystemMemory() - 4) {
        printf("Not enough GPU memory for this number of passwords !\n");
        exit(1);
    }

    // We need to determine how many batch we'll do to hash all passwords
    // We need to compute the batch size as well
    auto numberOfPass = (double) ((double) memUsed / (double) freeMem);
    if (numberOfPass < 1) {
        printf("Number of passes : %d\n", 1);
        return 1;
    }

    numberOfPass += 0.5;

    int finalNumberOfPass = (int) numberOfPass;
    if ((finalNumberOfPass % 2) != 0) finalNumberOfPass++;

    printf("Number of passes : %d\n", finalNumberOfPass);

    return finalNumberOfPass;
}

__host__ int computeBatchSize(int numberOfPass, int passwordNumber) {
    // If we have less than 1 round then the batch size is the number of passwords
    if (numberOfPass > 1) return (passwordNumber / numberOfPass);
    else return passwordNumber;
}

__host__ void initEmptyArrays(Password **passwords, Digest **results, int passwordNumber) {
    handleCudaError(hipHostMalloc(passwords, passwordNumber * sizeof(Password), hipHostMallocDefault));
    handleCudaError(hipHostMalloc(results, passwordNumber * sizeof(Digest), hipHostMallocDefault));
}

__host__ void initArrays(Password **passwords, Digest **results, int passwordNumber) {
    generatePasswords(passwords, passwordNumber);
    handleCudaError(hipHostMalloc(results, passwordNumber * sizeof(Digest), hipHostMallocDefault));
}

__device__ __host__ void printDigest(Digest *dig) {
    // Iterate through every byte of the digest
    for (unsigned char byte : dig->bytes) {
        printf("%02X", byte); // %02X formats as uppercase hex with leading zeroes
    }

    //printf("\n");
}

__device__ __host__ void printPassword(Password *pwd) {
    // Iterate through every byte of the password
    for (unsigned char byte : pwd->bytes) {
        printf("%c", byte);
    }
    //printf("\n");
}

__host__ void createFile(char *path, bool debug) {
    std::ofstream file(path);
    if (debug) printf("\nNew file created: %s.\n", path);
}

__host__ std::ofstream openFile(const char *path) {
    std::ofstream file;
    file.open(path);

    // Check if the file was correctly opened
    if (!file.is_open()) {
        printf("Error: couldn't open file in %s.\n", path);
    }

    return file;
}

__host__ void writePoint(char *path, Password **passwords, int number, int t, bool debug) {
    std::ofstream file = openFile(path);

    file << number << std::endl;
    file << PASSWORD_LENGTH << std::endl;
    file << t << std::endl;

    // Iterate through every point
    for (int i = 0; i < number; i++) {
        for(int j=0; j < PASSWORD_LENGTH; j++) {
            file << (*passwords)[i].bytes[j];
        }
        file << std::endl;
    }

    if (debug) printf("The point file was written.\n\n");
    file.close();
}


__host__ void writeEndingReduction(char *path, Password **passwords, Digest **results, int endNumber, bool debug) {
    std::ofstream file = openFile(path);

    // Iterate through every end point
    for (int i = 0; i < endNumber; i++) {
        file << (*passwords)[i].bytes << "-->";
        // Iterate through every byte of the end point
        for (int j = 0; j < HASH_LENGTH; j++) {
            char buf[HASH_LENGTH];
            sprintf(buf, "%02X", (*results)[i].bytes[j]); // %02X formats as uppercase hex with leading zeroes
            file << buf;
        }
        file << std::endl;
    }

    if (debug) printf("The end point reduction file was written.\n");
    file.close();
}

__host__ int computeT(int goRam, int mt) {
    double mtMax;

    // Recommended value
    double r = 19.83;

    // Need to compute mtMax first
    mtMax = (double)mt / (double)(1/(double)(1+(double)(1/r)));

    double domain = pow(62, PASSWORD_LENGTH);

    // Compute t knowing mtMax
    int result = (int)((2*domain) / (int)mtMax) - 2;
    if (result < 1) return 1;
    else return result;
}

__host__ int getM0(int goRam, int mt) {
    double mZero;
    double mtMax;
    double tmpMZero;

    // Recommended value
    double r = 19.83;

    // Choosing m0 based on host memory
    mZero = getNumberPassword(goRam);

    // Need to compute mtMax first
    mtMax = (double)mt / (double)(1/(double)(1+(double)(1/r)));

    tmpMZero = r * mtMax;

    if (tmpMZero > mZero && pow(62, PASSWORD_LENGTH) > tmpMZero) {
        printf("Chosen mt require a bigger m0 than memory available!\n");
        exit(1);
    }else {
        printf("m0: %d\n", (int)mZero);
        return (int)mZero;
    }
}

__host__ int getNumberPassword(int goRam) {

    double domain = pow(62, PASSWORD_LENGTH);

    size_t memLine = sizeof(Password) + sizeof(Digest);

    // memUsed = memLine * nbLine -> nbLine = memUsed / memLine
    // memUsed = totalMem - 4 Go
    // totalMem * 1000000000 pour passer de Giga octets à  octets

    long memUsed = ((long)goRam * (long)1000000000) - ((long)2 * (long)1000000000);

    int result = (int)pow(2, (int)log2((int)((long)memUsed / (int)memLine)));

    if (result > domain){
        result = (int)pow(2, ceil((int)log((int)domain)/log(2)));
    }

    printf("M0: %d\n", result);
    return result;
}

__host__ int getTotalSystemMemory() {
    long pages = sysconf(_SC_PHYS_PAGES);
    long page_size = sysconf(_SC_PAGE_SIZE);
    double value = ((double)(pages * page_size) / 1000000000) - 2;
    if (value > 31.0) return 32;
    else if (value > 15.0) return 16;
    else if (value > 7.0) return 8;
}