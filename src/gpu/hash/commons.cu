#include "hip/hip_runtime.h"
#include "commons.cuh"

__host__ Password * generatePasswords(int passwordNumber) {

    auto * result = (Password*) malloc(passwordNumber*sizeof(Password));

    char charSet[62] = {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x',
                      'y','z','1','2','3','4','5','6','7','8','9','0','A','B','C','D','E','F','G','H','I','J','K'
    ,'L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
    std::random_device rd; // obtain a random number from hardware
    std::mt19937 gen(rd()); // seed the generator
    std::uniform_int_distribution<> distr(0, 61); // define the range

    printf("\n==========GENERATING PASSWORDS==========\n");
    // Generate all passwords
    for(int j=0; j<passwordNumber; j++) {
        auto * currentPassword = (Password *) malloc(sizeof(Password));
        // Generate one password
        for (unsigned char &byte: (*currentPassword).bytes) {
            byte = charSet[distr(gen)];
        }

        result[j] = *(currentPassword);
        free(currentPassword);
    }
    printf("DONE, %d PASSWORDS GENERATED\n", passwordNumber);
    printf("====================\n");

    return result;
}

// Returns the number of batch that we need to do
__host__ double memoryAnalysis(int passwordNumber) {

    printf("\n==========MEMORY ANALYSIS==========\n");

    // Checking if THREAD_PER_BLOCK is a power of 2 because we will have memory problems otherwise
    if ((ceil(log2(THREAD_PER_BLOCK)) != floor(log2(THREAD_PER_BLOCK)))){
        printf("THREAD PER BLOCK VALUE IS NOT A POWER OF 2 !\n");
        exit(1);
    }

    size_t freeMem;
    size_t totalMem;
    hipError_t mem = hipMemGetInfo(&freeMem, &totalMem);

    // Just to keep a little of memory, just in case
    freeMem -= 500000000;

    // Checking errors on memory detection
    if (mem != hipSuccess) {
        printf("memory check failed with error \"%s\".\n",
               hipGetErrorString(mem));
        exit(1);
    }

    printf("MEMORY AVAILABLE : %ld Megabytes\n", (freeMem / 1000000));

    // Computing memory used by password and result array
    size_t memResult = sizeof(Digest) * passwordNumber;
    size_t memPasswords = sizeof(Password) * passwordNumber;
    size_t memUsed = memPasswords + memResult;

    printf("MEMORY USED BY RESULT ARRAY : %ld Megabytes\n",
           (memResult / 1000000));
    printf("MEMORY USED BY PASSWORD ARRAY : %ld Megabytes\n",
           (memPasswords / 1000000));

    printf("THIS MUCH MEMORY WILL BE USED : %ld Megabytes\n",
           (memUsed / 1000000));

    // We need to determine how many batch we'll do to hash all passwords
    // We need to compute the batch size as well
    auto numberOfPass = (double)((double)memUsed / (double)freeMem);

    printf("NUMBER OF PASS : %f\n", numberOfPass);

    printf("====================\n");

    return numberOfPass;
}

__host__ int computeBatchSize(double initialNumberOfPass, int passwordNumber) {
    int batchSize;

    // Formula to round down is : result = ((number + multiple/2) / multiple) *
    // multiple;
    initialNumberOfPass += 0.5;
    int numberOfPass = (int)initialNumberOfPass;
    if ((numberOfPass % 2) != 0) numberOfPass++;
    printf("%d\n", (int)numberOfPass);


    if ((int)numberOfPass > 1)
        batchSize = ((int)(passwordNumber / (int)numberOfPass));
    // If we have less than 1 round then the batch size is the number of
    // passwords
    else
        batchSize = passwordNumber;

    return batchSize;
}

__host__ void kernel(const double numberOfPass, int batchSize,
                     float *milliseconds, const clock_t *program_start,
                     Digest **h_results, Password **h_passwords, int passwordNumber) {

    printf("\n==========LAUNCHING KERNEL==========\n");

    *h_results = (Digest *)malloc(passwordNumber * sizeof(Digest));

    // Device copies
    Digest *d_results;
    Password *d_passwords;

    // Measure GPU time
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    int passwordRemaining = passwordNumber;
    int currentIndex = 0;

    printf("FIRST BATCH SIZE : %d\n", batchSize);

    // Main loop, we add +1 to be sure to do all the batches in case
    // we have 2.5 for example, it'll be 3 passes
    for (int i = 0; i < (int)numberOfPass + 1; i++) {
        // Temporary variable to measure GPU time inside this loop
        float tempMilli = 0;

        // GPU Malloc for the password array, size is batchSize
        hipMalloc(&d_passwords, sizeof(Password) * batchSize);
        hipMalloc(&d_results, sizeof(Digest) * batchSize);

        // If the currentIndex to save result is greater than the number of
        // password we must stop
        if (currentIndex >= passwordNumber) break;

        // If we have less than batchSize password to hash, then hash them all
        // but modify the batchSize to avoid index errors
        if (passwordRemaining <= batchSize) batchSize = passwordRemaining;
        printf("BATCHSIZE: %d\n", batchSize);

        // Debug print
        // printf("PASSWORD REMAINING : %d, BATCH SIZE : %d\n",
        // passwordRemaining, batchSize); printf("CURRENT INDEX : %d\n",
        // currentIndex);

        Password *source = *h_passwords;
        // Device copies
        hipMemcpy(d_passwords, &(source[currentIndex]), sizeof(Password) * batchSize,
                   hipMemcpyHostToDevice);

        hipEventRecord(start);
        ntlm_kernel<<<((batchSize) / THREAD_PER_BLOCK), THREAD_PER_BLOCK>>>(
            d_passwords, d_results);
        hipEventRecord(end);
        hipEventSynchronize(end);
        // Necessary procedure to record time and store the elasped time in
        // tempMilli
        hipEventElapsedTime(&tempMilli, start, end);
        *milliseconds += tempMilli;
        hipEventDestroy(start);
        hipEventDestroy(end);

        printf("KERNEL #%d DONE @ %f seconds\n", i,
               (double)(clock() - *program_start) / CLOCKS_PER_SEC);

        // Check for errors during kernel execution
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) {
            printf("kernel launch failed with error \"%s\".\n",
                   hipGetErrorString(cudaerr));
            exit(1);
        }

        Digest *destination = *h_results;
        // Device to host copy
        hipMemcpy(&(destination[currentIndex]), d_results,
                   sizeof(Digest) * batchSize, hipMemcpyDeviceToHost);

        // Fix the index because array begin at index 0, not 1
        if (i == 0) currentIndex += batchSize - 1;
        // If we don't have to fix it then just add batchSize
        else
            currentIndex += batchSize;
        passwordRemaining -= batchSize;

        printf("CURRENT INDEX: %d\n", currentIndex);
        // Debug
        // printf("NEW CURRENT INDEX : %d\n", currentIndex);

        // Cleanup before next loop to free memory
        hipFree(d_passwords);
        hipFree(d_results);
    }
    printf("====================\n");
}