#include "hip/hip_runtime.h"
#include "commons.cuh"

// Returns the number of batch that we need to do
__host__ double memoryAnalysis(int passwordNumber) {

    printf("\n==========MEMORY ANALYSIS==========\n");
    // Checking available memory on the device, store free memory into freeMem
    // and total memory into totalMem
    size_t freeMem;
    size_t totalMem;
    hipError_t mem = hipMemGetInfo(&freeMem, &totalMem);

    // Just to keep a little of memory, just in case
    freeMem -= 500000000;

    // Checking errors on memory detection
    if (mem != hipSuccess) {
        printf("memory check failed with error \"%s\".\n",
               hipGetErrorString(mem));
        exit(1);
    }

    printf("MEMORY AVAILABLE : %ld Megabytes\n", (freeMem / 1000000));

    // Computing memory used by password and result array
    size_t memResult = sizeof(Digest) * passwordNumber;
    size_t memPasswords = sizeof(Password) * passwordNumber;
    size_t memUsed = memPasswords + memResult;

    printf("MEMORY USED BY RESULT ARRAY : %ld Megabytes\n",
           (memResult / 1000000));
    printf("MEMORY USED BY PASSWORD ARRAY : %ld Megabytes\n",
           (memPasswords / 1000000));

    printf("THIS MUCH MEMORY WILL BE USED : %ld Megabytes\n",
           (memUsed / 1000000));

    // We need to determine how many batch we'll do to hash all passwords
    // We need to compute the batch size as well
    auto numberOfPass = (double)((double)memUsed / (double)freeMem);

    printf("NUMBER OF PASS : %f\n", numberOfPass);

    printf("====================\n");

    return numberOfPass;
}

__host__ int computeBatchSize(double numberOfPass, int passwordNumber) {
    int batchSize;

    // Formula to round down is : result = ((number + multiple/2) / multiple) *
    // multiple;
    if (numberOfPass > 1)
        batchSize = ((((int)(passwordNumber / numberOfPass)) + 1) / 2) * 2;

    // If we have less than 1 round then the batch size is the number of
    // passwords
    else
        batchSize = passwordNumber;

    return batchSize;
}

__host__ void kernel(const double numberOfPass, int batchSize,
                     float *milliseconds, const clock_t *program_start,
                     Digest **h_results, Password **h_passwords, int passwordNumber) {

    printf("\n==========LAUNCHING KERNEL==========\n");

    *h_results = (Digest *)malloc(passwordNumber * sizeof(Digest));

    // Device copies
    Digest *d_results;
    Password *d_passwords;

    // Measure GPU time
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    int passwordRemaining = passwordNumber;
    int currentIndex = 0;

    printf("FIRST BATCH SIZE : %d\n", batchSize);

    // Main loop, we add +1 to be sure to do all the batches in case
    // we have 2.5 for example, it'll be 3 passes
    for (int i = 0; i < (int)numberOfPass + 1; i++) {
        // Temporary variable to measure GPU time inside this loop
        float tempMilli = 0;

        // GPU Malloc for the password array, size is batchSize
        hipMalloc(&d_passwords, sizeof(Password) * batchSize);
        hipMalloc(&d_results, sizeof(Digest) * batchSize);

        // If the currentIndex to save result is greater than the number of
        // password we must stop
        if (currentIndex >= passwordNumber) break;

        // If we have less than batchSize password to hash, then hash them all
        // but modify the batchSize to avoid index errors
        if (passwordRemaining < batchSize) batchSize = passwordRemaining;

        // Debug print
        // printf("PASSWORD REMAINING : %d, BATCH SIZE : %d\n",
        // passwordRemaining, batchSize); printf("CURRENT INDEX : %d\n",
        // currentIndex);

        Password *source = *h_passwords;
        // Device copies
        hipMemcpy(d_passwords, &(source[currentIndex]), sizeof(Password) * batchSize,
                   hipMemcpyHostToDevice);

        hipEventRecord(start);
        ntlm_kernel<<<batchSize / THREAD_PER_BLOCK, THREAD_PER_BLOCK>>>(
            d_passwords, d_results);
        hipEventRecord(end);

        // Necessary procedure to record time and store the elasped time in
        // tempMilli
        hipEventSynchronize(end);
        hipEventElapsedTime(&tempMilli, start, end);
        *milliseconds += tempMilli;

        printf("KERNEL #%d DONE @ %f seconds\n", i,
               (double)(clock() - *program_start) / CLOCKS_PER_SEC);

        // Check for errors during kernel execution
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) {
            printf("kernel launch failed with error \"%s\".\n",
                   hipGetErrorString(cudaerr));
            exit(1);
        }

        Digest *destination = *h_results;
        // Device to host copy
        hipMemcpy(&(destination[currentIndex]), d_results,
                   sizeof(Digest) * batchSize, hipMemcpyDeviceToHost);

        // Fix the index because array begin at index 0, not 1
        if (i == 0) currentIndex += batchSize - 1;
        // If we don't have to fix it then just add batchSize
        else
            currentIndex += batchSize;
        passwordRemaining -= batchSize;

        // Debug
        // printf("NEW CURRENT INDEX : %d\n", currentIndex);

        // Cleanup before next loop to free memory
        hipFree(d_passwords);
        hipFree(d_results);
    }
}