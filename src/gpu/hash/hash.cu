#include "hip/hip_runtime.h"
#include "hash.cuh"

void hash(Password * h_passwords, Digest * h_results, int passwordNumber, int numberOfPass) {

    int batchSize = computeBatchSize(numberOfPass, passwordNumber);

    // Measure global time
    double program_time_used;
    clock_t program_start, program_end;
    program_start = clock();

    float milliseconds = 0;

    hashKernel(numberOfPass, batchSize, &milliseconds, &program_start, &h_results, &h_passwords, passwordNumber,
               THREAD_PER_BLOCK);

    // Compute GPU time and hash rate
    printf("GPU PARALLEL HASH TIME : %f milliseconds\n", milliseconds);
    printf("HASH RATE : %f MH/s\n",
           (passwordNumber / (milliseconds / 1000)) / 1000000);

    // End and compute total time
    program_end = clock();
    program_time_used =
            ((double) (program_end - program_start)) / CLOCKS_PER_SEC;
    printf("TOTAL EXECUTION TIME : %f seconds\n", program_time_used);
}


// Another version using a time variable, so we can retrieve its value
void hashTime(Password *h_passwords, Digest * h_results, int passwordNumber, float *milliseconds,
              int threadPerBlock, int numberOfPass) {

    int batchSize = computeBatchSize(numberOfPass, passwordNumber);

    // Measure global time
    double program_time_used;
    clock_t program_start, program_end;
    program_start = clock();

    hashKernel(numberOfPass, batchSize, milliseconds, &program_start, &h_results, &h_passwords, passwordNumber,
               threadPerBlock);

    // Compute GPU time and hash rate
    printf("GPU PARALLEL HASH TIME : %f milliseconds\n", *milliseconds);
    printf("HASH RATE : %f MH/s\n",
           (passwordNumber / (*milliseconds / 1000)) / 1000000);

    // End and compute total time
    program_end = clock();
    program_time_used =
            ((double) (program_end - program_start)) / CLOCKS_PER_SEC;
    printf("TOTAL EXECUTION TIME : %f seconds\n", program_time_used);
}
