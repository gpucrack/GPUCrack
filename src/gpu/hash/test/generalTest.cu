#include "generalTest.cuh"

int main() {

    int passwordNumber = DEFAULT_PASSWORD_NUMBER;

    // Simulate when we send password as input
    Password * passwords = generatePasswords(passwordNumber);

    Digest * result;

    hipError_t status = hipHostMalloc(&result, passwordNumber * sizeof(Digest));
    if (status != hipSuccess)
        printf("Error allocating pinned host memory\n");

    auto numberOfPass = memoryAnalysis(passwordNumber);

    parallelized_hash(passwords, result, passwordNumber, numberOfPass);

    hipHostFree(passwords);
    hipHostFree(result);

    return (0);
}