#include "hashGeneralTest.cuh"

int main() {

    int passwordNumber = getNumberPassword(6);

    Password * passwords;
    Digest * result;

    initArrays(&passwords, &result, passwordNumber);

    auto numberOfPass = memoryAnalysisGPU(passwordNumber);

    hash(passwords, result, passwordNumber, numberOfPass, false);

    //generateNewPasswords(&passwords, passwordNumber);

    //hash(passwords, result, passwordNumber, numberOfPass, false);

    printPassword(&passwords[0]);
    printf(" --> ");
    printDigest(&result[0]);
    printf("\n");
    printPassword(&passwords[1]);
    printf(" --> ");
    printDigest(&result[1]);
    printf("\n...\n");
    printPassword(&passwords[passwordNumber-2]);
    printf(" --> ");
    printDigest(&result[passwordNumber-2]);
    printf("\n");
    printPassword(&passwords[passwordNumber-1]);
    printf(" --> ");
    printDigest(&result[passwordNumber-1]);
    printf("\n");

    hipHostFree(passwords);
    hipHostFree(result);

    return (0);
}