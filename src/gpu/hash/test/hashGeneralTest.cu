#include "hashGeneralTest.cuh"

int main() {

    int passwordNumber = getNumberPassword(8);

    Password * passwords;
    Digest * result;

    initArrays(&passwords, &result, passwordNumber);

    auto numberOfPass = memoryAnalysis(passwordNumber);

    hash(passwords, result, passwordNumber, numberOfPass, false);

    //generateNewPasswords(&passwords, passwordNumber);

    //hash(passwords, result, passwordNumber, numberOfPass, false);

    hipHostFree(passwords);
    hipHostFree(result);

    return (0);
}