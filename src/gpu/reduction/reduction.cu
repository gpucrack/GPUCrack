#include "hip/hip_runtime.h"
#include "reduction.cuh"

// The character set used for passwords. We declare it in the host scope and in the device scope.
// The character set used for passwords.
__device__ static const unsigned char charset[CHARSET_LENGTH] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 'A', 'B', 'C', 'D', 'E',
                                                      'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'R', 'S', 'T',
                                                      'U', 'V', 'W', 'X', 'Y', 'Z', 'a', 'b', 'c',
                                                      'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r',
                                                      's', 't',
                                                      'u', 'v', 'w', 'x', 'y', 'z'};

// The character set used for digests (NTLM hashes).
static const unsigned char hashset[DIGEST_CHARSET_LENGTH] = {0x88, 0x46, 0xF7, 0xEA, 0xEE, 0x8F, 0xB1,
                                                             0x17, 0xAD, 0x06, 0xBD, 0xD8, 0x30, 0xB7,
                                                             0x58, 0x6C};

void generate_digests_random(Digest **digests, int n) {
    for (int j = 0; j < n; j++) {
        for (int i = HASH_LENGTH - 1; i >= 0; i--) {
            (*digests)[j].bytes[i] = hashset[rand() % CHARSET_LENGTH];
        }
    }
}

__host__ __device__ void reduceDigest(unsigned int pos, Digest *digest, Password *plain_text, unsigned long domain) {

    // index so that we are inside the right domain
    unsigned long index = ((*digest).value + pos) % domain;

    for(int i=PASSWORD_LENGTH-1; i>=0; i--){
        (*plain_text).bytes[i] = charset[index % (unsigned long)CHARSET_LENGTH];

        // Dividing by index, so we lose a power each time to stay in the correct domain for the next character
        index /= CHARSET_LENGTH;
    }

    /*
    for(int i=0; i<PASSWORD_LENGTH; i++){
        (*plain_text).bytes[i] = charset[((*digest).bytes[i] + index) % CHARSET_LENGTH];
    }*/
}

__global__ void reduceDigests(Digest *digests, Password *plain_texts, int column, unsigned long domain) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    reduceDigest(column, &digests[idx], &plain_texts[idx], domain);
}

int count_duplicates(Password **passwords, bool debug, int passwordNumber) {
    int count = 0;
    for (int i = 0; i < passwordNumber; i++) {
        if (debug) printf("Searching for duplicate of password number %d...\n", i);
        for (int j = i + 1; j < passwordNumber; j++) {
            // Increment count by 1 if duplicate found
            if (memcmp((*passwords)[i].bytes, (*passwords)[j].bytes, PASSWORD_LENGTH) != 0) {
                printf("Found a duplicate : ");
                printPassword(&(*passwords)[i]);
                count++;
            }
        }
    }
    return count;
}

void display_reductions(Digest *digests, Password *passwords, int n) {
    for (int i = 0; i < n; i++) {
        printDigest(&(digests[i]));
        printf(" --> ");
        printPassword(&(passwords[i]));
        printf("\n");
    }
}

__host__ void reduceKernel(int passwordNumber, int numberOfPass, int batchSize, float *milliseconds,
                          Password **h_passwords, Digest **h_results, int threadPerBlock) {
    Password * d_passwords;
    Digest * d_results;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    int reductionRemaining = passwordNumber;
    int currentIndex = 0;

    unsigned long domain = pow(CHARSET_LENGTH, PASSWORD_LENGTH);

    // Main loop, we add +1 to be sure to do all the batches in case
    // we have 2.5 for example, it'll be 3 passes
    for (int i = 0; i < numberOfPass; i++) {
        // Temporary variable to measure GPU time inside this loop
        float tempMilli = 0;

        // If the currentIndex to save result is greater than the number of
        // password we must stop
        if (currentIndex >= passwordNumber) break;

        // If we have less than batchSize password to hash, then hash them all
        // but modify the batchSize to avoid index errors
        if (reductionRemaining <= batchSize) batchSize = reductionRemaining;

        // GPU Malloc for the password array, size is batchSize
        hipMalloc(&d_passwords, sizeof(Password) * batchSize);
        hipMalloc(&d_results, sizeof(Digest) * batchSize);

        Digest *source = *h_results;

        // Device copies
        hipMemcpy(d_results, &(source[currentIndex]), sizeof(Digest) * batchSize,
                        hipMemcpyHostToDevice);

        hipEventRecord(start);
        // Reduce all those digests into passwords
        reduceDigests<<<((batchSize) / threadPerBlock), threadPerBlock>>>(d_results,
                                                                          d_passwords, 1, domain);

        hipEventRecord(end);
        hipEventSynchronize(end);

        // Necessary procedure to record time and store the elasped time in
        // tempMilli
        hipEventElapsedTime(&tempMilli, start, end);
        *milliseconds += tempMilli;
        hipEventDestroy(start);
        hipEventDestroy(end);

        // Check for errors during hashKernel execution
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) {
            printf("hashKernel launch failed with error \"%s\".\n",
                   hipGetErrorString(cudaerr));
            exit(1);
        }

        Password *destination2 = *h_passwords;
        // Device to host copy
        hipMemcpy(&(destination2[currentIndex]), d_passwords,
                        sizeof(Password) * batchSize, hipMemcpyDeviceToHost);

        currentIndex += batchSize;
        reductionRemaining -= batchSize;

        // Cleanup before next loop to free memory
        hipFree(d_passwords);
        hipFree(d_results);
    }
}

__host__ void
reduce(Password *h_passwords, Digest *h_results, int passwordNumber, int numberOfPass, int threadsPerBlock) {
    double program_time_used;
    clock_t program_start, program_end;
    program_start = clock();

    // Generate DEFAULT_PASSWORD_NUMBER digests
    printf("Generating digests...\n");
    generate_digests_random(&h_results, passwordNumber);
    printf("Digest generation done!\n");

    float milliseconds = 0;

    int batchSize = computeBatchSize(numberOfPass, passwordNumber);

    reduceKernel(passwordNumber, numberOfPass, batchSize, &milliseconds, &h_passwords, &h_results, threadsPerBlock);

    //display_reductions(h_results, h_passwords, passwordNumber);

    printf("TOTAL GPU TIME : %f milliseconds\n", milliseconds);

    double reduce_rate = ((double)passwordNumber / (milliseconds / 1000)) / 1000000;

    printf("Reduction of %d digests ended after %f milliseconds.\n Reduction rate: %f MR/s.\n", passwordNumber,
           (double) milliseconds, reduce_rate);

    /*
    int dup = count_duplicates(&h_passwords, false, 0);
    printf("Found %d duplicate(s) among the %d reduced passwords (%f percent).\n", dup, passwordNumber,
           ((double) dup / passwordNumber) * 100);*/

    program_end = clock();
    program_time_used =
            ((double) (program_end - program_start)) / CLOCKS_PER_SEC;
    printf("TOTAL EXECUTION TIME : %f seconds\n", program_time_used);
}