#include "hip/hip_runtime.h"
#include "reduction.cuh"

// Global variables for GPU
__device__ int password_length_gpu = PASSWORD_LENGTH;
__device__ int charset_length_gpu = CHARSET_LENGTH;

// The character set used for passwords. We declare it in the host scope and in the device scope.
// The character set used for passwords.
__device__ static const unsigned char charset[CHARSET_LENGTH] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 'A', 'B', 'C', 'D', 'E',
                                                      'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'R', 'S', 'T',
                                                      'U', 'V', 'W', 'X', 'Y', 'Z', 'a', 'b', 'c',
                                                      'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r',
                                                      's', 't',
                                                      'u', 'v', 'w', 'x', 'y', 'z', '-', '_'};

// The character set used for digests (NTLM hashes).
static const unsigned char hashset[DIGEST_CHARSET_LENGTH] = {0x88, 0x46, 0xF7, 0xEA, 0xEE, 0x8F, 0xB1,
                                                             0x17, 0xAD, 0x06, 0xBD, 0xD8, 0x30, 0xB7,
                                                             0x58, 0x6C};

void generate_digests_random(Digest **digests, int n) {
    for (int j = 0; j < n; j++) {
        for (int i = HASH_LENGTH - 1; i >= 0; i--) {
            (*digests)[j].bytes[i] = hashset[rand() % CHARSET_LENGTH];
        }
    }
}

__device__ void reduce_digest2(unsigned long index, Digest * digest, Password * plain_text) {
    for (int i = 0; i < CEILING(HASH_LENGTH,4) - 1; i++) {
        (*plain_text).i[i] = charset[((*digest).i[i] + index) % 64];
    }
}

__global__ void reduce_digests2(Digest *digests, Password *plain_texts) {
    unsigned long idx = blockIdx.x * blockDim.x + threadIdx.x;
    reduce_digest2(idx, digests, plain_texts);
}

inline int pwdcmp(Password &p1, Password &p2) {
    for (int i = 0; i < CEILING(PASSWORD_LENGTH, 4); i++) {
        if (p1.i[i] != p2.i[i]) {
            return false;
        }
    }
    return true;
}

int count_duplicates(Password **passwords, bool debug, int passwordNumber) {
    int count = 0;
    for (int i = 0; i < passwordNumber; i++) {
        if (debug) printf("Searching for duplicate of password number %d...\n", i);
        for (int j = i + 1; j < passwordNumber; j++) {
            // Increment count by 1 if duplicate found
            if (memcmp((*passwords)[i].bytes, (*passwords)[j].bytes, PASSWORD_LENGTH) != 0) {
                printf("Found a duplicate : ");
                printPassword(&(*passwords)[i]);
                count++;
            }
        }
    }
    return count;
}

void display_reductions(Digest **digests, Password **passwords, int n) {
    for (int i = 0; i < n; i++) {
        printDigest(&(*digests)[i]);
        printf(" --> ");
        printPassword(&(*passwords)[i]);
        printf("\n");
    }
}

__host__ void reduceKernel(int passwordNumber, int numberOfPass, int batchSize, float *milliseconds,
                          Password **h_passwords, Digest **h_results, int threadPerBlock) {
    Password * d_passwords;
    Digest * d_results;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    int reductionRemaining = passwordNumber;
    int currentIndex = 0;

    // Main loop, we add +1 to be sure to do all the batches in case
    // we have 2.5 for example, it'll be 3 passes
    for (int i = 0; i < numberOfPass; i++) {
        // Temporary variable to measure GPU time inside this loop
        float tempMilli = 0;

        // If the currentIndex to save result is greater than the number of
        // password we must stop
        if (currentIndex >= passwordNumber) break;

        // If we have less than batchSize password to hash, then hash them all
        // but modify the batchSize to avoid index errors
        if (reductionRemaining <= batchSize) batchSize = reductionRemaining;

        // GPU Malloc for the password array, size is batchSize
        hipMalloc(&d_passwords, sizeof(Password) * batchSize);
        hipMalloc(&d_results, sizeof(Digest) * batchSize);

        Password *source = *h_passwords;

        // Device copies
        hipMemcpy(d_passwords, &(source[currentIndex]), sizeof(Password) * batchSize,
                        hipMemcpyHostToDevice);

        hipEventRecord(start);
        // Reduce all those digests into passwords
        reduce_digests2<<<((batchSize) / threadPerBlock), threadPerBlock>>>(d_results, d_passwords);

        hipEventRecord(end);
        hipEventSynchronize(end);

        // Necessary procedure to record time and store the elasped time in
        // tempMilli
        hipEventElapsedTime(&tempMilli, start, end);
        *milliseconds += tempMilli;
        hipEventDestroy(start);
        hipEventDestroy(end);

        // Check for errors during hashKernel execution
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) {
            printf("hashKernel launch failed with error \"%s\".\n",
                   hipGetErrorString(cudaerr));
            exit(1);
        }

        Password *destination2 = *h_passwords;
        // Device to host copy

        hipMemcpy(&(destination2[currentIndex]), d_passwords,
                        sizeof(Password) * batchSize, hipMemcpyDeviceToHost);
        currentIndex += batchSize;
        reductionRemaining -= batchSize;

        // Cleanup before next loop to free memory
        hipFree(d_passwords);
        hipFree(d_results);
    }
}

__host__ void
reduce(Password *h_passwords, Digest *h_results, int passwordNumber, int numberOfPass, int threadsPerBlock) {
    double program_time_used;
    clock_t program_start, program_end;
    program_start = clock();

    // Generate DEFAULT_PASSWORD_NUMBER digests
    printf("Generating digests...\n");
    generate_digests_random(&h_results, passwordNumber);
    printf("Digest generation done!\n");

    float milliseconds = 0;

    int batchSize = computeBatchSize(numberOfPass, passwordNumber);

    reduceKernel(passwordNumber, numberOfPass, batchSize, &milliseconds, &h_passwords, &h_results, threadsPerBlock);

    printf("TOTAL GPU TIME : %f milliseconds\n", milliseconds);

    double reduce_rate = (passwordNumber / (milliseconds / 1000)) / 1000000;

    printf("Reduction of %d digests ended after %f milliseconds.\n Reduction rate: %f MR/s.\n", passwordNumber,
           (double) milliseconds, reduce_rate);

    int dup = count_duplicates(&h_passwords, false, 0);
    printf("Found %d duplicate(s) among the %d reduced passwords (%f percent).\n", dup, passwordNumber,
           ((double) dup / passwordNumber) * 100);

    program_end = clock();
    program_time_used =
            ((double) (program_end - program_start)) / CLOCKS_PER_SEC;
    printf("TOTAL EXECUTION TIME : %f seconds\n", program_time_used);
}