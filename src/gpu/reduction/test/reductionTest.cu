#include "reductionTest.cuh"

int main() {
    int passwordNumber = getNumberPassword(8);

    Password * passwords;
    Digest * result;

    initEmptyArrays(&passwords, &result, passwordNumber);

    auto numberOfPass = memoryAnalysis(passwordNumber);

    reduce(passwords, result, passwordNumber, numberOfPass, THREAD_PER_BLOCK);

    for(int i=0; i<10; i++) {
        printPassword(&passwords[i]);
        printf("\n");
        reduceDigest(0, &result[i], &passwords[i], 916132832);
        printf("CPU: ");
        printPassword(&passwords[i]);
        printf("\n");
    }

    hipHostFree(passwords);
    hipHostFree(result);

    return 0;
}