#include "reductionTest.cuh"

int main() {
    int passwordNumber = getNumberPassword(8);

    Password * passwords;
    Digest * result;

    initEmptyArrays(&passwords, &result, passwordNumber);

    auto numberOfPass = memoryAnalysis(passwordNumber);

    reduce(passwords, result, passwordNumber, numberOfPass, THREAD_PER_BLOCK);

    hipHostFree(passwords);
    hipHostFree(result);

    return 0;
}