#include "chainBenchmarkTest.cuh"

int main() {
    int passwordNumber = getNumberPassword(getTotalSystemMemory());

    Password * passwords;
    Digest * result;

    initArrays(&passwords, &result, passwordNumber);

    auto numberOfPass = memoryAnalysis(passwordNumber);

    int numberOfColumn = 1000;

    generateChains(passwords, result, passwordNumber,
                   numberOfPass, numberOfColumn, false, THREAD_PER_BLOCK, true, false);

    hipHostFree(passwords);
    hipHostFree(result);

    return 0;
}