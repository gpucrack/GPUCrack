#include "chainBenchmarkTest.cuh"

int main(){
    int passwordNumber = getNumberPassword(8);

    double maxHashRedRate = 0;
    double bestHashRedRateMean = 0;
    int bestValue = 0;

    Password * passwords;
    Digest * result;

    initArrays(&passwords, &result, passwordNumber);

    auto numberOfPass = memoryAnalysis(passwordNumber);

    // int numberOfColumn = computeT(16);
    int numberOfColumn = 100;

    float milliseconds = 0;
    float tempMilli = 0;

    for (int k = 2; k <= MAX_THREAD_NUMBER; k = k * 2) {

        double hashRedRateSum = 0;

        for (int i = 0; i < NUMBER_OF_TEST; i++) {

            // Measure GPU time
            hipEvent_t start, end;
            hipEventCreate(&start);
            hipEventCreate(&end);

            hipEventRecord(start);
            generateChains(passwords, result, passwordNumber,
                           numberOfPass, numberOfColumn, false, k);
            hipEventRecord(end);
            hipEventSynchronize(end);

            hipEventElapsedTime(&tempMilli, start, end);
            milliseconds += tempMilli;
            hipEventDestroy(start);
            hipEventDestroy(end);

            double hashRedRate = (((float) (passwordNumber) / (tempMilli / 1000)) / 1000000) * (float) numberOfColumn;

            if (hashRedRate > maxHashRedRate) {
                maxHashRedRate = hashRedRate;
            }

            hashRedRateSum += hashRedRate;

        }

        double currentHashRateMean = hashRedRateSum / NUMBER_OF_TEST;
        if (currentHashRateMean > bestHashRedRateMean) {
            bestValue = k;
            bestHashRedRateMean = currentHashRateMean;
        }

    }

    printf("MAX HASHREDRATE : %f\n", maxHashRedRate);
    printf("BEST THREAD PER BLOCK VALUE : %d WITH MEAN : %f\n", bestValue, bestHashRedRateMean);
    printf("NOTE: THIS HASHREDRATE IS LOWER THAN THE REAL ONE\n");

    hipHostFree(passwords);
    hipHostFree(result);

    return 0;
}