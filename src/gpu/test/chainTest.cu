#include "chainTest.cuh"

int main(){
    int passwordNumber = 1073741824;

    Password * passwords;
    Digest * result;

    initArrays(&passwords, &result, passwordNumber);

    auto numberOfPass = memoryAnalysis(passwordNumber);

    generateChains(passwords, result, passwordNumber, numberOfPass);

    hipHostFree(passwords);
    hipHostFree(result);

    return 0;
}
