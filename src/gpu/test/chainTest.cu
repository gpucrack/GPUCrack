#include "chainTest.cuh"

int main(){
    int passwordNumber = DEFAULT_PASSWORD_NUMBER;

    Password * passwords;
    Digest * result;

    initArrays(&passwords, &result, passwordNumber);

    auto numberOfPass = memoryAnalysis(passwordNumber);

    generateChains(passwords, result, passwordNumber, numberOfPass);

    hipHostFree(passwords);
    hipHostFree(result);

    return 0;
}
