#include "hip/hip_runtime.h"
#include "chainTest.cuh"

int main(int argc, char *argv[]){
    char *start_path;
    char *end_path;
    int pwd_length = atoi(argv[1]);

    long domain = pow(CHARSET_LENGTH, pwd_length);

    long idealM0 = (long)(0.1*(double)domain);

    long idealMtMax = (long)((double)((double)idealM0/(double)19.83));

    long mtMax = getNumberPassword(atoi(argv[2]), pwd_length);

    mtMax = idealMtMax;

    long passwordNumber = idealM0;
    //long passwordNumber = 18980;

    int t = computeT(mtMax, pwd_length);
    //int t = 500;

    //mtMax = 949;

    printf("mtMax: %ld\n", mtMax);

    printf("m0: %ld\n", passwordNumber);

    printf("Password length: %d\n", pwd_length);
    printf("Number of columns (t): %d\n\n", t);

    Password * passwords;
    Digest * result;

    // check
    initArrays(&passwords, &result, passwordNumber);

    auto numberOfPass = memoryAnalysisGPU(passwordNumber);

    passwords[0].bytes[0] = 'Z';
    passwords[0].bytes[1] = 'm';
    passwords[0].bytes[2] = 'd';

    // Adjust t depending on the chain length you want to test
    generateChains(passwords, passwordNumber, numberOfPass, t,
                   false, THREAD_PER_BLOCK, true, true, result, PASSWORD_LENGTH, start_path, end_path);

    printf("Should be first password inside endpoints:\n");
    printPassword(&passwords[0]);
    printf("\n");


    hipHostFree(passwords);
    hipHostFree(result);

    return 0;
}
