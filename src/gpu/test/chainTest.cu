#include "chainTest.cuh"

int main(){
    int passwordNumber = 1048576;

    Password * passwords;
    Digest * result;

    initArrays(&passwords, &result, passwordNumber);

    auto numberOfPass = memoryAnalysis(passwordNumber);

    generateChains(passwords, result, passwordNumber, numberOfPass);

    hipHostFree(passwords);
    hipHostFree(result);

    return 0;
}
