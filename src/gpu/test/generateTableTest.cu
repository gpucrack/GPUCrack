#include "generateTableTest.cuh"

int main(int argc, char *argv[]){
    if (argc != 2) {
        printf("Error: not enough arguments given.\n Usage: 'generateTable mt', where mt is the desired number of end points.");
        exit(1);
    }

    printSignature();

    int passwordNumber = getM0(getTotalSystemMemory(), atoi(argv[1]));

    Password * passwords;
    Digest * result;

    initArrays(&passwords, &result, passwordNumber);

    auto numberOfPass = memoryAnalysis(passwordNumber);

    //int t = computeT(getTotalSystemMemory(), atoi(argv[1]));
    int t = 100;

    printf("Number of columns: %d\n\n", t);

    generateChains(passwords, result, passwordNumber, numberOfPass, t,
                   true, THREAD_PER_BLOCK, false);

    printf("Chains generated!\n");

    hipHostFree(passwords);
    hipHostFree(result);

    return 0;
}
